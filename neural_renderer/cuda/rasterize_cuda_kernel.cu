#include "hip/hip_runtime.h"
#include <iostream>
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// for the older gpus atomicAdd with double arguments does not exist
#if  __CUDA_ARCH__ < 600 and defined(__CUDA_ARCH__)
static __inline__ __device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN) } while (assumed != old);
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

// implementation of atomicExch for double input
// adapted from https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
__device__ double atomicExch(double* address, double val) {
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val));
    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN) } while (assumed != old);

    } while (assumed != old);
    return __longlong_as_double(old);
}

namespace{
template <typename scalar_t>
__global__ void forward_face_index_map_cuda_kernel(
        const scalar_t* __restrict__ faces,
        int32_t*  face_index_map,
        scalar_t* weight_map,
        scalar_t*  depth_map,
        scalar_t* face_inv_map,
        int32_t* lock,
        size_t batch_size,
        size_t num_faces,
        int image_size,
        scalar_t near,
        scalar_t far,
        int return_rgb,
        int return_alpha,
        int return_depth) {
    /* batch number, face, number, image size, face[v012][RGB] */
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_faces * batch_size) {
        return;
    }
    const int bn = i / num_faces;
    const int fn = i % num_faces;
    const int is = image_size;
    const scalar_t* face = &faces[i * 9];
    
    /* return if backside */
    if ((face[7] - face[1]) * (face[3] - face[0]) < (face[4] - face[1]) * (face[6] - face[0]))
        return;
    
    /* pi[0], pi[1], pi[2] = leftmost, middle, rightmost points */
    int pi[3];
    if (face[0] < face[3]) {
        if (face[6] < face[0]) pi[0] = 2; else pi[0] = 0;
        if (face[3] < face[6]) pi[2] = 2; else pi[2] = 1;
    } else {
        if (face[6] < face[3]) pi[0] = 2; else pi[0] = 1;
        if (face[0] < face[6]) pi[2] = 2; else pi[2] = 0;
    }
    for (int k = 0; k < 3; k++)
        if (pi[0] != k && pi[2] != k)
            pi[1] = k;
    
    /* p[num][xyz]: x, y is normalized from [-1, 1] to [0, is - 1]. */
    scalar_t p[3][3];
    for (int num = 0; num < 3; num++) {
        for (int dim = 0; dim < 3; dim++) {
            if (dim != 2) {
                p[num][dim] = 0.5 * (face[3 * pi[num] + dim] * is + is - 1);
            } else {
                p[num][dim] = face[3 * pi[num] + dim];
            }
        }
    }
    if (p[0][0] == p[2][0])
        return; // line, not triangle 
    
    /* compute face_inv */
    scalar_t face_inv[9] = {
        p[1][1] - p[2][1], p[2][0] - p[1][0], p[1][0] * p[2][1] - p[2][0] * p[1][1],
        p[2][1] - p[0][1], p[0][0] - p[2][0], p[2][0] * p[0][1] - p[0][0] * p[2][1],
        p[0][1] - p[1][1], p[1][0] - p[0][0], p[0][0] * p[1][1] - p[1][0] * p[0][1]};
    scalar_t face_inv_denominator = (
        p[2][0] * (p[0][1] - p[1][1]) +
        p[0][0] * (p[1][1] - p[2][1]) +
        p[1][0] * (p[2][1] - p[0][1]));
    for (int k = 0; k < 9; k++)
        face_inv[k] /= face_inv_denominator;
    
    /* from left to right */
    // const int xi_min = min(max(ceil(p[0][0]), 0.), is - 1.);
    // const int xi_max = max(min(p[2][0], is - 1.), 0.);
    const int xi_min = max(ceil(p[0][0]), 0.);
    const int xi_max = min(p[2][0], is - 1.);
    for (int xi = xi_min; xi <= xi_max; xi++) {
        /* compute yi_min and yi_max */
        scalar_t yi1, yi2;
        if (xi <= p[1][0]) {
            if (p[1][0] - p[0][0] != 0) {
                yi1 = (p[1][1] - p[0][1]) / (p[1][0] - p[0][0]) * (xi - p[0][0]) + p[0][1];
            }
            else {
                yi1 = p[1][1];
            }
        }
        else {
            if (p[2][0] - p[1][0] != 0) {
                yi1 = (p[2][1] - p[1][1]) / (p[2][0] - p[1][0]) * (xi - p[1][0]) + p[1][1];
            }
            else {
                yi1 = p[1][1];
            }
        }
        yi2 = (p[2][1] - p[0][1]) / (p[2][0] - p[0][0]) * (xi - p[0][0]) + p[0][1];
    
        /* from up to bottom */
        int yi_min = max(0., ceil(min(yi1, yi2)));
        int yi_max = min(max(yi1, yi2), is - 1.);
        for (int yi = yi_min; yi <= yi_max; yi++) {
            /* index in output buffers */
            int index = bn * is * is + yi * is + xi;
            // remove it after debugging
            if (index > batch_size * is * is -1)
                continue;
    
            /* compute w = face_inv * p */
            scalar_t w[3];
            for (int k = 0; k < 3; k++)
                w[k] = face_inv[3 * k + 0] * xi + face_inv[3 * k + 1] * yi + face_inv[3 * k + 2];
    
            /* sum(w) -> 1, 0 < w < 1 */
            scalar_t w_sum = 0;
            for (int k = 0; k < 3; k++) {
                w[k] = min(max(w[k], 0.), 1.);
                w_sum += w[k];
            }
            for (int k = 0; k < 3; k++)
                w[k] /= w_sum;
    
            /* compute 1 / zp = sum(w / z) */
            const scalar_t zp = 1. / (w[0] / p[0][2] + w[1] / p[1][2] + w[2] / p[2][2]);
            // index = 2;
            if (zp <= near || far <= zp)
                continue;
    
            /* lock and update */
            bool locked = false;
            do {
                if (locked = atomicCAS(&lock[index], 0, 1) == 0) {
                    if (zp < atomicAdd(&depth_map[index], 0)) {
                         size_t record = 0;
                         depth_map[index] = zp;
                         face_index_map[index] = fn;
                         for (int k = 0; k < 3; k++)
                             atomicExch(&weight_map[3 * index + pi[k]], w[k]);
                         if (return_depth) {
                             for (int k = 0; k < 3; k++)
                                 for (int l = 0; l < 3; l++)
                                    atomicExch(
                                        &face_inv_map[9 * index + 3 * pi[l] + k], face_inv[3 * l + k]);
                         }
                    }
                    atomicExch(&lock[index], 0);
                }
            } while (!locked);
        }
    }
}

template <typename scalar_t>
__global__ void forward_texture_sampling_cuda_kernel(
		const scalar_t* faces,
		const scalar_t* textures,
		const int32_t* face_index_map,
		const scalar_t* weight_map,
		const scalar_t* depth_map,
		scalar_t* rgb_map,
		int32_t* sampling_index_map,
        scalar_t* sampling_weight_map,
        size_t batch_size,
        int num_faces,
        int image_size,
        int texture_size,
        scalar_t eps) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch_size * image_size * image_size) {
        return;
    }
    const int face_index = face_index_map[i];
    
    if (face_index >= 0) {
        /*
            from global variables:
            batch number, num of faces, image_size, face[v012][RGB], pixel[RGB], weight[v012],
            texture[ts][ts][ts][RGB], sampling indices[8], sampling_weights[8];
        */
        const int bn = i / (image_size * image_size);
        const int nf = num_faces;
        const int ts = texture_size;
        const scalar_t* face = &faces[face_index * 9];
        const scalar_t* texture = &textures[(bn * nf + face_index) * ts * ts * ts * 3];
        scalar_t* pixel = &rgb_map[i * 3];
        const scalar_t* weight = &weight_map[i * 3];
        const scalar_t depth = depth_map[i];
        int32_t* sampling_indices = &sampling_index_map[i * 8];
        scalar_t* sampling_weights = &sampling_weight_map[i * 8];
    
        /* get texture index (float) */
        scalar_t texture_index_float[3];
        for (int k = 0; k < 3; k++) { scalar_t tif = weight[k] * (ts - 1) * (depth / (face[3 * k + 2]));
            tif = max(tif, 0.);
            tif = min(tif, ts - 1 - eps);
            texture_index_float[k] = tif;
        }
    
        /* blend */
        scalar_t new_pixel[3] = {0, 0, 0};
        for (int pn = 0; pn < 8; pn++) {
            scalar_t w = 1;                         // weight
            int texture_index_int[3];            // index in source (int)
            for (int k = 0; k < 3; k++) {
                if ((pn >> k) % 2 == 0) {
                    w *= 1 - (texture_index_float[k] - (int)texture_index_float[k]);
                    texture_index_int[k] = (int)texture_index_float[k];
                }
                else {
                    w *= texture_index_float[k] - (int)texture_index_float[k];
                    texture_index_int[k] = (int)texture_index_float[k] + 1;
                }
            }
    
            int isc = texture_index_int[0] * ts * ts + texture_index_int[1] * ts + texture_index_int[2];
            for (int k = 0; k < 3; k++)
                new_pixel[k] += w * texture[isc * 3 + k];
            sampling_indices[pn] = isc;
            sampling_weights[pn] = w;
        }
        for (int k = 0; k < 3; k++)
            pixel[k] = new_pixel[k];
    }
}

template <typename scalar_t>
__global__ void backward_pixel_map_cuda_kernel(
		const scalar_t* faces,
        int32_t*  face_index_map,
        scalar_t*  rgb_map,
        scalar_t*  alpha_map,
        scalar_t*  grad_rgb_map,
        scalar_t*  grad_alpha_map,
        scalar_t*  grad_faces,
        size_t batch_size,
        size_t num_faces,
        int image_size,
        scalar_t eps,
        int return_rgb,
        int return_alpha) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch_size * num_faces) {
        return;
    }
    const int bn = i / num_faces;
    const int fn = i % num_faces;
    const int is = image_size;
    const scalar_t* face = &faces[i * 9];
    scalar_t grad_face[9] = {};

    /* check backside */
    if ((face[7] - face[1]) * (face[3] - face[0]) < (face[4] - face[1]) * (face[6] - face[0])) return;

    /* for each edge */
    for (int edge_num = 0; edge_num < 3; edge_num++) {
        /* set points of target edge */
        int pi[3];
        scalar_t pp[3][2];
        for (int num = 0; num < 3; num++)
            pi[num] = (edge_num + num) % 3;
        for (int num = 0; num < 3; num++) {
            for (int dim = 0; dim < 2; dim++) {
                pp[num][dim] = 0.5 * (face[3 * pi[num] + dim] * is + is - 1);
            }
        }

        /* for dy, dx */
        for (int axis = 0; axis < 2; axis++) {
            /* */
            scalar_t p[3][2];
            for (int num = 0; num < 3; num++) {
                for (int dim = 0; dim < 2; dim++) {
                    p[num][dim] = pp[num][(dim + axis) % 2];
                }
            }

            /* set direction */
            int direction;
            if (axis == 0) {
                if (p[0][0] < p[1][0])
                    direction = -1;
                else
                    direction = 1;
            } else {
                if (p[0][0] < p[1][0])
                    direction = 1;
                else
                    direction = -1;
            }

            /* along edge */
            int d0_from, d0_to;
            d0_from = max(ceil(min(p[0][0], p[1][0])), 0.);
            d0_to = min(max(p[0][0], p[1][0]), is - 1.);
            for (int d0 = d0_from; d0 <= d0_to; d0++) {
                /* get cross point */
                int d1_in, d1_out;
                const scalar_t d1_cross = (p[1][1] - p[0][1]) / (p[1][0] - p[0][0]) * (d0 - p[0][0]) + p[0][1];
                if (0 < direction)
                    d1_in = floor(d1_cross);
                else
                    d1_in = ceil(d1_cross);
                d1_out = d1_in + direction;

                /* continue if cross point is not shown */
                if (d1_in < 0 || is <= d1_in)
                    continue;
                if (d1_out < 0 || is <= d1_out)
                    continue;

                /* get color of in-pixel and out-pixel */
                scalar_t alpha_in;
                scalar_t alpha_out;
                scalar_t *rgb_in;
                scalar_t *rgb_out;
                int map_index_in, map_index_out;
                if (axis == 0) {
                    map_index_in = bn * is * is + d1_in * is + d0;
                    map_index_out = bn * is * is + d1_out * is + d0;
                }
                else {
                    map_index_in = bn * is * is + d0 * is + d1_in;
                    map_index_out = bn * is * is + d0 * is + d1_out;
                }
                if (return_alpha) {
                    alpha_in = alpha_map[map_index_in];
                    alpha_out = alpha_map[map_index_out];
                }
                if (return_rgb) {
                    rgb_in = &rgb_map[map_index_in * 3];
                    rgb_out = &rgb_map[map_index_out * 3];
                }

                /* out */
                bool is_in_fn = (face_index_map[map_index_in] == fn);
                if (is_in_fn) {
                    int d1_limit;
                    if (0 < direction)
                        d1_limit = is - 1;
                    else
                        d1_limit = 0;
                    int d1_from = max(min(d1_out, d1_limit), 0);
                    int d1_to = min(max(d1_out, d1_limit), is - 1);
                    scalar_t* alpha_map_p;
                    scalar_t* grad_alpha_map_p;
                    scalar_t* rgb_map_p;
                    scalar_t* grad_rgb_map_p;
                    int map_offset, map_index_from;
                    if (axis == 0) {
                        map_offset = is;
                        map_index_from = bn * is * is + d1_from * is + d0;
                    }
                    else {
                        map_offset = 1;
                        map_index_from = bn * is * is + d0 * is + d1_from;
                    }
                    if (return_alpha) {
                        alpha_map_p = &alpha_map[map_index_from];
                        grad_alpha_map_p = &grad_alpha_map[map_index_from];
                    }
                    if (return_rgb) {
                        rgb_map_p = &rgb_map[map_index_from * 3];
                        grad_rgb_map_p = &grad_rgb_map[map_index_from * 3];
                    }
                    for (int d1 = d1_from; d1 <= d1_to; d1++) {
                        scalar_t diff_grad = 0;
                        if (return_alpha) {
                            diff_grad += (*alpha_map_p - alpha_in) * *grad_alpha_map_p;
                        }
                        if (return_rgb) {
                            for (int k = 0; k < 3; k++)
                                diff_grad += (rgb_map_p[k] - rgb_in[k]) * grad_rgb_map_p[k];
                        }
                        if (return_alpha) {
                            alpha_map_p += map_offset;
                            grad_alpha_map_p += map_offset;
                        }
                        if (return_rgb) {
                            rgb_map_p += 3 * map_offset;
                            grad_rgb_map_p += 3 * map_offset;
                        }
                        if (diff_grad <= 0)
                            continue;
                        if (p[1][0] != d0) {
                            scalar_t dist = (p[1][0] - p[0][0]) / (p[1][0] - d0) * (d1 - d1_cross) * 2. / is;
                            dist = (0 < dist) ? dist + eps : dist - eps;
                            grad_face[pi[0] * 3 + (1 - axis)] -= diff_grad / dist;
                        }
                        if (p[0][0] != d0) {
                            scalar_t dist = (p[1][0] - p[0][0]) / (d0 - p[0][0]) * (d1 - d1_cross) * 2. / is;
                            dist = (0 < dist) ? dist + eps : dist - eps;
                            grad_face[pi[1] * 3 + (1 - axis)] -= diff_grad / dist;
                        }
                    }
                }

                /* in */
                {
                    int d1_limit;
                    scalar_t d0_cross2;
                    if ((d0 - p[0][0]) * (d0 - p[2][0]) < 0) {
                        d0_cross2 = (p[2][1] - p[0][1]) / (p[2][0] - p[0][0]) * (d0 - p[0][0]) + p[0][1];
                    }
                    else {
                        d0_cross2 = (p[1][1] - p[2][1]) / (p[1][0] - p[2][0]) * (d0 - p[2][0]) + p[2][1];
                    }
                    if (0 < direction)
                        d1_limit = ceil(d0_cross2);
                    else
                        d1_limit = floor(d0_cross2);
                    int d1_from = max(min(d1_in, d1_limit), 0);
                    int d1_to = min(max(d1_in, d1_limit), is - 1);

                    int* face_index_map_p;
                    scalar_t* alpha_map_p;
                    scalar_t* grad_alpha_map_p;
                    scalar_t* rgb_map_p;
                    scalar_t* grad_rgb_map_p;
                    int map_index_from;
                    int map_offset;
                    if (axis == 0)
                        map_offset = is;
                    else
                        map_offset = 1;
                    if (axis == 0) {
                        map_index_from = bn * is * is + d1_from * is + d0;
                    }
                    else {
                        map_index_from = bn * is * is + d0 * is + d1_from;
                    }
                    face_index_map_p = &face_index_map[map_index_from] - map_offset;
                    if (return_alpha) {
                        alpha_map_p = &alpha_map[map_index_from] - map_offset;
                        grad_alpha_map_p = &grad_alpha_map[map_index_from] - map_offset;
                    }
                    if (return_rgb) {
                        rgb_map_p = &rgb_map[map_index_from * 3] - 3 * map_offset;
                        grad_rgb_map_p = &grad_rgb_map[map_index_from * 3] - 3 * map_offset;
                    }

                    for (int d1 = d1_from; d1 <= d1_to; d1++) {
                        face_index_map_p += map_offset;
                        if (return_alpha) {
                            alpha_map_p += map_offset;
                            grad_alpha_map_p += map_offset;
                        }
                        if (return_rgb) {
                            rgb_map_p += 3 * map_offset;
                            grad_rgb_map_p += 3 * map_offset;
                        }
                        if (*face_index_map_p != fn)
                            continue;

                        scalar_t diff_grad = 0;
                        if (return_alpha) {
                            diff_grad += (*alpha_map_p - alpha_out) * *grad_alpha_map_p;
                        }
                        if (return_rgb) {
                            for (int k = 0; k < 3; k++)
                                diff_grad += (rgb_map_p[k] - rgb_out[k]) * grad_rgb_map_p[k];
                        }
                        if (diff_grad <= 0)
                            continue;

                        if (p[1][0] != d0) {
                            scalar_t dist = (p[1][0] - p[0][0]) / (p[1][0] - d0) * (d1 - d1_cross) * 2. / is;
                            dist = (0 < dist) ? dist + eps : dist - eps;
                            grad_face[pi[0] * 3 + (1 - axis)] -= diff_grad / dist;
                        }
                        if (p[0][0] != d0) {
                            scalar_t dist = (p[1][0] - p[0][0]) / (d0 - p[0][0]) * (d1 - d1_cross) * 2. / is;
                            dist = (0 < dist) ? dist + eps : dist - eps;
                            grad_face[pi[1] * 3 + (1 - axis)] -= diff_grad / dist;
                        }
                    }
                }
            }
        }
    }

    /* set to global gradient variable */
    for (int k = 0; k < 9; k++)
        grad_faces[i * 9 + k] = grad_face[k];
}

template <typename scalar_t>
__global__ void backward_textures_cuda_kernel(
        const int32_t* face_index_map,
        scalar_t* sampling_weight_map,
        int32_t* sampling_index_map,
        scalar_t* grad_rgb_map,
        scalar_t* grad_textures,
        size_t batch_size,
        size_t num_faces,
        int image_size,
        size_t texture_size) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch_size * image_size * image_size) {
        return;
    }
    const int face_index = face_index_map[i];
    if (0 <= face_index) {
        int is = image_size;
        int nf = num_faces;
        int ts = texture_size;
        int bn = i / (is * is);    // batch number [0 -> bs]
    
        scalar_t* grad_texture = &grad_textures[(bn * nf + face_index) * ts * ts * ts * 3];
        scalar_t* sampling_weight_map_p = &sampling_weight_map[i * 8];
        int* sampling_index_map_p = &sampling_index_map[i * 8];
        for (int pn = 0; pn < 8; pn++) {
            scalar_t w = *sampling_weight_map_p++;
            int isc = *sampling_index_map_p++;
            scalar_t* grad_texture_p = &grad_texture[isc * 3];
            scalar_t* grad_rgb_map_p = &grad_rgb_map[i * 3];
            for (int k = 0; k < 3; k++)
                atomicAdd(grad_texture_p++, w * *grad_rgb_map_p++);
        }
    }
}

template <typename scalar_t>
__global__ void backward_depth_map_cuda_kernel(
        const scalar_t* __restrict__ faces,
        const scalar_t* __restrict__ depth_map,
        const int32_t* __restrict__ face_index_map,
        const scalar_t* __restrict__ face_inv_map,
        const scalar_t* __restrict__ weight_map,
        scalar_t* __restrict__ grad_depth_map,
        scalar_t* __restrict__ grad_faces,
        size_t batch_size,
        size_t num_faces,
        int image_size) {
    
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch_size * image_size * image_size) {
        return;
    }
    const int fn = face_index_map[i];
    if (0 <= fn) {
        const int nf = num_faces;
        const int is = image_size;
        const int bn = i / (is * is);
        const scalar_t* face = &faces[(bn * nf + fn) * 9];
        const scalar_t depth = depth_map[i];
        const scalar_t depth2 = depth * depth;
        const scalar_t* face_inv = &face_inv_map[i * 9];
        const scalar_t* weight = &weight_map[i * 3];
        const scalar_t grad_depth = grad_depth_map[i];
        scalar_t* grad_face = &grad_faces[(bn * nf + fn) * 9];
    
        /* derivative wrt z */
        for (int k = 0; k < 3; k++) {
            const scalar_t z_k = face[3 * k + 2];
            atomicAdd(&grad_face[3 * k + 2], grad_depth * weight[k] * depth2 / (z_k * z_k));
        }
    
        /* derivative wrt x, y */
        scalar_t tmp[3] = {};
        for (int k = 0; k < 3; k++) {
            for (int l = 0; l < 3; l++) {
                tmp[k] += -face_inv[3 * l + k] / face[3 * l + 2];
            }
        }
        for (int k = 0; k < 3; k++) {
            for (int l = 0; l < 2; l++) {
            // k: point number, l: dimension
            atomicAdd(&grad_face[3 * k + l], -grad_depth * tmp[l] * weight[k] * depth2 * is / 2);
            }
        }
    }
}
}

std::vector<at::Tensor> forward_face_index_map_cuda(
        at::Tensor faces,
        at::Tensor face_index_map,
        at::Tensor weight_map,
        at::Tensor depth_map,
        at::Tensor face_inv_map,
        at::Tensor lock,
        int image_size,
        float near,
        float far,
        int return_rgb,
        int return_alpha,
        int return_depth) {

    const auto batch_size = faces.size(0);
    const auto num_faces = faces.size(1);
    const int threads = 1024;
    const int blocks = (batch_size * num_faces - 1) / threads +1;

    AT_DISPATCH_FLOATING_TYPES(faces.type(), "forward_face_index_map_cuda", ([&] {
      forward_face_index_map_cuda_kernel<scalar_t><<<blocks, threads>>>(
          faces.data<scalar_t>(),
          face_index_map.data<int32_t>(),
          weight_map.data<scalar_t>(),
          depth_map.data<scalar_t>(),
          face_inv_map.data<scalar_t>(),
          lock.data<int32_t>(),
          batch_size,
          num_faces,
          image_size,
          (scalar_t) near,
          (scalar_t) far,
          return_rgb,
          return_alpha,
          return_depth);
      }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
            printf("Error in forward_face_index_map: %s\n", hipGetErrorString(err));
    return {face_index_map, weight_map, depth_map, face_inv_map};
}

std::vector<at::Tensor> forward_texture_sampling_cuda(
        at::Tensor faces,
        at::Tensor textures,
        at::Tensor face_index_map,
        at::Tensor weight_map,
        at::Tensor depth_map,
        at::Tensor rgb_map,
        at::Tensor sampling_index_map,
        at::Tensor sampling_weight_map,
        int image_size,
        float eps) {

    const auto batch_size = faces.size(0);
    const auto num_faces = faces.size(1);
    const auto texture_size = textures.size(2);
    const int threads = 1024;
    const int blocks = (batch_size * image_size * image_size - 1) / threads + 1;

    AT_DISPATCH_FLOATING_TYPES(faces.type(), "forward_texture_sampling_cuda", ([&] {
      forward_texture_sampling_cuda_kernel<scalar_t><<<blocks, threads>>>(
          faces.data<scalar_t>(),
          textures.data<scalar_t>(),
          face_index_map.data<int32_t>(),
          weight_map.data<scalar_t>(),
          depth_map.data<scalar_t>(),
          rgb_map.data<scalar_t>(),
		  sampling_index_map.data<int32_t>(),
		  sampling_weight_map.data<scalar_t>(),
          batch_size,
		  num_faces,
          image_size,
          texture_size,
          eps);
      }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
            printf("Error in forward_texture_sampling: %s\n", hipGetErrorString(err));

    return {rgb_map, sampling_index_map, sampling_weight_map};
}

at::Tensor backward_pixel_map_cuda(
        at::Tensor faces,
        at::Tensor face_index_map,
        at::Tensor rgb_map,
        at::Tensor alpha_map,
        at::Tensor grad_rgb_map,
        at::Tensor grad_alpha_map,
        at::Tensor grad_faces,
        int image_size,
        float eps,
        int return_rgb,
        int return_alpha) {
    
    const auto batch_size = faces.size(0);
    const auto num_faces = faces.size(1);
    const int threads = 1024;
    const int blocks = (batch_size * num_faces - 1) / threads + 1;

    AT_DISPATCH_FLOATING_TYPES(faces.type(), "backward_pixel_map_cuda", ([&] {
      backward_pixel_map_cuda_kernel<scalar_t><<<blocks, threads>>>(
          faces.data<scalar_t>(),
          face_index_map.data<int32_t>(),
          rgb_map.data<scalar_t>(),
          alpha_map.data<scalar_t>(),
          grad_rgb_map.data<scalar_t>(),
          grad_alpha_map.data<scalar_t>(),
          grad_faces.data<scalar_t>(),
          batch_size,
		  num_faces,
          image_size,
          (scalar_t) eps,
          return_rgb,
          return_alpha);
      }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
            printf("Error in backward_pixel_map: %s\n", hipGetErrorString(err));

    return grad_faces;
}

at::Tensor backward_textures_cuda(
        at::Tensor face_index_map,
        at::Tensor sampling_weight_map,
        at::Tensor sampling_index_map,
        at::Tensor grad_rgb_map,
        at::Tensor grad_textures,
        int num_faces) {

    const auto batch_size = face_index_map.size(0);
    const auto image_size = face_index_map.size(1);
    const auto texture_size = grad_textures.size(2);
    const int threads = 1024;
    const int blocks = (batch_size * image_size * image_size - 1) / threads + 1;

    AT_DISPATCH_FLOATING_TYPES(sampling_weight_map.type(), "backward_textures_cuda", ([&] {
      backward_textures_cuda_kernel<scalar_t><<<blocks, threads>>>(
          face_index_map.data<int32_t>(),
          sampling_weight_map.data<scalar_t>(),
          sampling_index_map.data<int32_t>(),
          grad_rgb_map.data<scalar_t>(),
          grad_textures.data<scalar_t>(),
          batch_size,
          num_faces,
          image_size,
          texture_size);
      }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
            printf("Error in backward_textures: %s\n", hipGetErrorString(err));

    return grad_textures;
}
at::Tensor backward_depth_map_cuda(
        at::Tensor faces,
        at::Tensor depth_map,
        at::Tensor face_index_map,
        at::Tensor face_inv_map,
        at::Tensor weight_map,
        at::Tensor grad_depth_map,
        at::Tensor grad_faces,
        int image_size) {

    const auto batch_size = faces.size(0);
    const auto num_faces = faces.size(1);
    const int threads = 1024;
    const int blocks = (batch_size * image_size * image_size - 1) / threads + 1;

    AT_DISPATCH_FLOATING_TYPES(faces.type(), "backward_depth_map_cuda", ([&] {
      backward_depth_map_cuda_kernel<scalar_t><<<blocks, threads>>>(
          faces.data<scalar_t>(),
          depth_map.data<scalar_t>(),
          face_index_map.data<int32_t>(),
          face_inv_map.data<scalar_t>(),
          weight_map.data<scalar_t>(),
          grad_depth_map.data<scalar_t>(),
          grad_faces.data<scalar_t>(),
          batch_size,
          num_faces,
          image_size);
      }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
            printf("Error in backward_depth_map: %s\n", hipGetErrorString(err));

    return grad_faces;
}
